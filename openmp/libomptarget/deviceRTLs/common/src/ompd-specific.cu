
#include <hip/hip_runtime.h>
#ifdef OMPD_SUPPORT
#include "../ompd-specific.h"
#include "../omptarget.h"
/**
   * Declaration of symbols to hold struct size and member offset information
    */

__device__ __shared__ static int ompd_target_initialized;

#define ompd_target_declare_access(t,m) __device__ __shared__ uint64_t ompd_access__##t##__##m##_;
OMPD_FOREACH_ACCESS(ompd_target_declare_access)
#undef ompd_target_declare_access

#define ompd_target_declare_sizeof_member(t,m) __device__ __shared__ uint64_t ompd_sizeof__##t##__##m##_;
    OMPD_FOREACH_ACCESS(ompd_target_declare_sizeof_member)
#undef ompd_target_declare_sizeof_member

#define ompd_target_declare_sizeof(t) __device__ __shared__ uint64_t ompd_sizeof__##t##_;
    OMPD_FOREACH_SIZEOF(ompd_target_declare_sizeof)
#undef ompd_target_declare_sizeof

__device__ void ompd_init ( void )
{
  if (ompd_target_initialized)
    return;

#define ompd_target_init_access(t,m) ompd_access__##t##__##m##_ = (uint64_t)&(((t*)0)->m);
  OMPD_FOREACH_ACCESS(ompd_target_init_access)
#undef ompd_target_init_access

#define ompd_target_init_sizeof_member(t,m) ompd_sizeof__##t##__##m##_ = sizeof(((t*)0)->m);
  OMPD_FOREACH_ACCESS(ompd_target_init_sizeof_member)
#undef ompd_target_init_sizeof_member

#define ompd_target_init_sizeof(t) ompd_sizeof__##t##_ = sizeof(t);
  OMPD_FOREACH_SIZEOF(ompd_target_init_sizeof)
#undef ompd_target_init_sizeof

  omptarget_nvptx_threadPrivateContext->ompd_levelZeroParallelInfo.level = 0;
  if (isSPMDMode()) {
    omptarget_nvptx_threadPrivateContext->teamContext.levelZeroTaskDescr
        .ompd_thread_info.enclosed_parallel.parallel_tasks =
            &omptarget_nvptx_threadPrivateContext->levelOneTaskDescr[0];
  } else {
    // generic mode
    omptarget_nvptx_threadPrivateContext->ompd_levelZeroParallelInfo
        .parallel_tasks = &omptarget_nvptx_threadPrivateContext->teamContext
            .levelZeroTaskDescr;
  }

  ompd_target_initialized = 1;
}

INLINE void ompd_init_thread(omptarget_nvptx_TaskDescr *currTaskDescr,
                             void *task_func, uint8_t implicit) {
  currTaskDescr->ompd_thread_info.blockIdx_x = blockIdx.x;
  currTaskDescr->ompd_thread_info.threadIdx_x = threadIdx.x;
  currTaskDescr->ompd_thread_info.task_function = task_func;
  currTaskDescr->ompd_thread_info.task_implicit = implicit;
}

__device__ void ompd_set_device_specific_thread_state(
    omptarget_nvptx_TaskDescr *taskDescr, omp_state_t state) {
    taskDescr->ompd_thread_info.state = state;
}

__device__ void  ompd_set_device_thread_state(omp_state_t state) {
  ompd_set_device_specific_thread_state(getMyTopTaskDescriptor(isSPMDMode()), state);
}

__device__ void ompd_init_thread_parallel() {
  omptarget_nvptx_TaskDescr *currTaskDescr = getMyTopTaskDescriptor(isSPMDMode());
  ompd_init_thread(currTaskDescr, omptarget_nvptx_workFn, 1);
  ompd_set_device_specific_thread_state(currTaskDescr, omp_state_work_parallel);
}

__device__ void ompd_init_thread_master() {
  omptarget_nvptx_TaskDescr *currTaskDescr = getMyTopTaskDescriptor(isSPMDMode());
  ompd_init_thread(currTaskDescr, NULL, 1);
  ompd_set_device_specific_thread_state(currTaskDescr, omp_state_work_serial);
}

__device__ void ompd_init_explicit_task(void *task_func) {
    omptarget_nvptx_TaskDescr *taskDescr = getMyTopTaskDescriptor(isSPMDMode());
    ompd_init_thread(taskDescr, task_func, 0);
}

__device__ void ompd_bp_parallel_begin (){ asm (""); }
__device__ void ompd_bp_parallel_end (){ asm (""); }
__device__ void ompd_bp_task_begin (){ asm (""); }
__device__ void ompd_bp_task_end (){ asm (""); }
__device__ void ompd_bp_thread_begin (){ asm (""); }
__device__ void ompd_bp_thread_end (){ asm (""); }
#endif /* OMPD_SUPPORT */
