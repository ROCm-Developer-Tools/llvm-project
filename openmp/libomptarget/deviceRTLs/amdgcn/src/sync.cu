#include "hip/hip_runtime.h"
//===------------ sync.h - NVPTX OpenMP synchronizations --------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// Include all synchronization.
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"
#include "target_impl.h"

////////////////////////////////////////////////////////////////////////////////
// KMP Ordered calls
////////////////////////////////////////////////////////////////////////////////

EXTERN void __kmpc_ordered(kmp_Ident *loc, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_ordered\n");
}

EXTERN void __kmpc_end_ordered(kmp_Ident *loc, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_end_ordered\n");
}

////////////////////////////////////////////////////////////////////////////////
// KMP Barriers
////////////////////////////////////////////////////////////////////////////////

// a team is a block: we can use CUDA native synchronization mechanism
// FIXME: what if not all threads (warps) participate to the barrier?
// We may need to implement it differently

EXTERN int32_t __kmpc_cancel_barrier(kmp_Ident *loc_ref, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_cancel_barrier\n");
  __kmpc_barrier(loc_ref, tid);
  PRINT0(LD_SYNC, "completed kmpc_cancel_barrier\n");
  return 0;
}

EXTERN void __kmpc_barrier(kmp_Ident *loc_ref, int32_t tid) {
  if (checkRuntimeUninitialized(loc_ref)) {
    ASSERT0(LT_FUSSY, checkSPMDMode(loc_ref),
            "Expected SPMD mode with uninitialized runtime.");
    __kmpc_barrier_simple_spmd(loc_ref, tid);
  } else {
    tid = GetLogicalThreadIdInBlock(checkSPMDMode(loc_ref));
    int numberOfActiveOMPThreads =
        GetNumberOfOmpThreads(checkSPMDMode(loc_ref));
    if (numberOfActiveOMPThreads > 1) {
      if (checkSPMDMode(loc_ref)) {
        __kmpc_barrier_simple_spmd(loc_ref, tid);
      } else {
        // The #threads parameter must be rounded up to the WARPSIZE.
        int threads =
            WARPSIZE * ((numberOfActiveOMPThreads + WARPSIZE - 1) / WARPSIZE);

        PRINT(LD_SYNC,
              "call kmpc_barrier with %d omp threads, sync parameter %d\n",
              (int)numberOfActiveOMPThreads, (int)threads);
        // Barrier #1 is for synchronization among active threads.
        named_sync(L1_BARRIER, threads);
      }
    } // numberOfActiveOMPThreads > 1
    PRINT0(LD_SYNC, "completed kmpc_barrier\n");
  }
}

// Emit a simple barrier call in SPMD mode.  Assumes the caller is in an L0
// parallel region and that all worker threads participate.
EXTERN void __kmpc_barrier_simple_spmd(kmp_Ident *loc_ref, int32_t tid) {
  PRINT0(LD_SYNC, "call kmpc_barrier_simple_spmd\n");
  __kmpc_impl_syncthreads();
  PRINT0(LD_SYNC, "completed kmpc_barrier_simple_spmd\n");
}

///////////////////////////////////////////////////////////////////////////////
/// The following four functions are used to coordinate barriers between
/// the master and worker warps in a generic kernel.  The amdgcn architecture
/// does not have a partial barrier so this coordination needs to be
/// recover if a user or implicit barrier is encountered in the sequential
/// part of the master warp or the parallel part of the worker warps.
///
/// __kmpc_amd_worker_start has a barrier to prevent worker warps
/// from starting till the master warp sets the workFn.  This function sets
/// workers_active to true after the barrier to keep the master warp
/// at its barrier in case a worker encounters an explicit or implicit
/// barrier.
EXTERN void __kmpc_amd_worker_start(kmp_Ident *loc_ref, int32_t tid) {
  PRINT0(LD_SYNC, "call kmpc_amd_worker_start\n");
  omptarget_workers_active = false;
  __kmpc_impl_syncthreads();
  while(omptarget_master_active) __kmpc_impl_syncthreads();
  omptarget_workers_active = true;
  PRINT0(LD_SYNC, "completed kmpc_amd_worker_start\n");
}

/// __kmpc_amd_worker_end sets workers_active to false and then
/// issues a barrier to release the master warp to terminate or get
/// the next subregion of work to process.
EXTERN void __kmpc_amd_worker_end(kmp_Ident *loc_ref, int32_t tid) {
  PRINT0(LD_SYNC, "call kmpc_amd_worker_end\n");
  omptarget_workers_active = false;
  __kmpc_impl_syncthreads(); // to sync with 2nd barrier in master_end
  PRINT0(LD_SYNC, "completed kmpc_amd_worker_end\n");
}

/// __kmpc_amd_master_start is executed first by the master.  It sets
/// master_active to true to keep the worker warps at its first barrier
/// in case the sequential part of the target region encounters any
/// barrier, explicit or otherwise.
EXTERN void __kmpc_amd_master_start(kmp_Ident *loc_ref, int32_t tid) {
  PRINT0(LD_SYNC, "call kmpc_amd_master_start\n");
  omptarget_master_active = true;
  PRINT0(LD_SYNC, "completed kmpc_amd_master_start\n");
}

/// __kmpc_amd_master_end sets master_active to false and then enters
/// the double barrier. The first releases the worker warps. The 2nd
/// barrier holds the master warp until the workers are done. If the worker
/// warps encounter a user barrier (implicitly or explicitly), the master
/// warp needs to loop at the barrier until it knows the worker is
/// really done by testing omptarget_workers_active.
EXTERN void __kmpc_amd_master_end(kmp_Ident *loc_ref, int32_t tid) {
  PRINT0(LD_SYNC, "call kmpc_amd_master_end\n");
  omptarget_master_active = false;
  __kmpc_impl_syncthreads();
  __kmpc_impl_syncthreads();
  while(omptarget_workers_active) __kmpc_impl_syncthreads();
  PRINT0(LD_SYNC, "completed kmpc_amd_master_end\n");
}

EXTERN void __kmpc_amd_master_terminate(kmp_Ident *loc_ref, int32_t tid) {
  PRINT0(LD_SYNC, "call kmpc_amd_master_terminate\n");
  omptarget_master_active = false;
  __kmpc_impl_syncthreads();
  PRINT0(LD_SYNC, "completed kmpc_amd_master_terminate\n");
}

// Emit a simple barrier call in Generic mode.  Assumes the caller is in an L0
// parallel region and that all worker threads participate.
EXTERN void __kmpc_barrier_simple_generic(kmp_Ident *loc_ref, int32_t tid) {
  int numberOfActiveOMPThreads = GetNumberOfThreadsInBlock() - WARPSIZE;
  // The #threads parameter must be rounded up to the WARPSIZE.
  int threads =
      WARPSIZE * ((numberOfActiveOMPThreads + WARPSIZE - 1) / WARPSIZE);

  PRINT(LD_SYNC,
        "call kmpc_barrier_simple_generic with %d omp threads, sync parameter "
        "%d\n",
        (int)numberOfActiveOMPThreads, (int)threads);
  // Barrier #1 is for synchronization among active threads.
  named_sync(L1_BARRIER, threads);
  PRINT0(LD_SYNC, "completed kmpc_barrier_simple_generic\n");
}

////////////////////////////////////////////////////////////////////////////////
// KMP MASTER
////////////////////////////////////////////////////////////////////////////////

EXTERN int32_t __kmpc_master(kmp_Ident *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_master\n");
  return IsTeamMaster(global_tid);
}

EXTERN void __kmpc_end_master(kmp_Ident *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_end_master\n");
  ASSERT0(LT_FUSSY, IsTeamMaster(global_tid), "expected only master here");
}

////////////////////////////////////////////////////////////////////////////////
// KMP SINGLE
////////////////////////////////////////////////////////////////////////////////

EXTERN int32_t __kmpc_single(kmp_Ident *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_single\n");
  // decide to implement single with master; master get the single
  return IsTeamMaster(global_tid);
}

EXTERN void __kmpc_end_single(kmp_Ident *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_end_single\n");
  // decide to implement single with master: master get the single
  ASSERT0(LT_FUSSY, IsTeamMaster(global_tid), "expected only master here");
  // sync barrier is explicitely called... so that is not a problem
}

////////////////////////////////////////////////////////////////////////////////
// Flush
////////////////////////////////////////////////////////////////////////////////

EXTERN void __kmpc_flush(kmp_Ident *loc) {
  PRINT0(LD_IO, "call kmpc_flush\n");
  __threadfence_system();
}

////////////////////////////////////////////////////////////////////////////////
// Vote
////////////////////////////////////////////////////////////////////////////////

#ifdef __AMDGCN__
EXTERN __kmpc_impl_lanemask_t __kmpc_warp_active_thread_mask64() {
  PRINT0(LD_IO, "call __kmpc_warp_active_thread_mask64\n");
  return __kmpc_impl_activemask();
}
#else
EXTERN __kmpc_impl_lanemask_t __kmpc_warp_active_thread_mask() {
  PRINT0(LD_IO, "call __kmpc_warp_active_thread_mask\n");
  return __kmpc_impl_activemask();
}
#endif
