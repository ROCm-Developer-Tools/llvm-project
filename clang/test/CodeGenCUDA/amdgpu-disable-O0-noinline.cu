#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -disable-O0-optnone -disable-O0-noinline -o - %s | FileCheck --check-prefix=CHECK %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-NOT: Function Attrs: {{.*}} optnone
// CHECK-NOT: Function Attrs: {{.*}} noinline

__device__ void foo() {
}

__global__ void bar() {
}
